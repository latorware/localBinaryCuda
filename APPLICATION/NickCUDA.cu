#include "hip/hip_runtime.h"
#include <iostream>
#include <QTextBrowser>
#include <hip/driver_types.h>
#include <hip/hip_runtime.h>
using namespace std;


#include "./Lib/stb_image_write.h"

#include <malloc.h>
#include <>


#define BLOCKSIZE 32 //32x32 = 1024 (max threads per block)

//max image size: 10.000 x 10.000 pixels
//__constant__ float GrayscaledImageConstant[10000 * 10000];


/*
// gpu kernel function
__global__ void test_kernel(void) {
}
*/

/*
extern "C"
void wrapper(QTextBrowser * outputDisplay)
{
	test_kernel << <1, 1 >> > ();
}
*/



__global__ void NickKernelMethod1(float* grayscaledImageDevice, float* FinalImageDevice, float k, int width, int height, int tamanyMEITATFinestra)
{
	//identificadors threads, fila i columna
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int row = (blockIdx.y * BLOCKSIZE) + ty;
	int col = (blockIdx.x * BLOCKSIZE) + tx;

	//bordes de la finestra lliscant
	int beginrow = max(0, row-tamanyMEITATFinestra);
	int begincolumn = max(0, col - tamanyMEITATFinestra);
	int endrow = min(height-1, row + tamanyMEITATFinestra); 
	int endcolumn = min(width-1, col + tamanyMEITATFinestra); 


	//calcular el pixel actual
	int numeropixelsfinestra = (endrow - beginrow + 1) * (endcolumn - begincolumn + 1);

	if (row < height && col < width)
	{
		float temp;
		float Total_sum = 0;
		float Total_sum_pow2 = 0;
		for (int i = beginrow; i <= endrow; i = i + 1)
			for (int j = begincolumn; j <= endcolumn; j = j + 1)
			{
				temp = grayscaledImageDevice[i * width + j]; 
				//printf("%f \n", temp);
				Total_sum = Total_sum + temp;
				Total_sum_pow2 = Total_sum_pow2 + (temp * temp);
			}

		//printf("%f \n", Total_sum);
		float mean = Total_sum / numeropixelsfinestra;
		float Threshold = mean + k * sqrtf((Total_sum_pow2 - mean * mean) / numeropixelsfinestra);
		//printf("%f      %f \n", Threshold, grayscaledImageDevice[row * width + col]);

		if (Threshold < grayscaledImageDevice[row * width + col])
		{
			FinalImageDevice[row * width + col] = 1; 
			//printf("Yes \n"); 
		}
		else
		{
			FinalImageDevice[row * width + col] = 0; 
			//printf("No \n"); 
		}
		//printf("%f \n", FinalImageDevice[row * width + col]);
	}


}

extern "C"
string NICKGPUMethod1(const float* grayscaledImage, int tamanyFinestra, float k, int width, int height, QTextBrowser * outputDisplay, string fileOUTGPUMETHOD1NICK)
{
	float millisecondsMemoryEvent = 0;
	float millisecondsKernelEvent = 0;
	float millisecondsMemoryBackEvent = 0;
	dim3 dimGrid, dimBlock; 

	dimBlock.x = BLOCKSIZE;
	dimBlock.y = BLOCKSIZE;
	dimBlock.z = 1;
	// + BLOCKSIZE necessari pels pixels que queden
	dimGrid.x = (width + BLOCKSIZE - 1) / BLOCKSIZE;
	dimGrid.y = (height + BLOCKSIZE - 1) / BLOCKSIZE;
	dimGrid.z = 1;

	float* FinalImageHost = (float*)malloc(width * height * sizeof(float));
	unsigned char* FinalImageHostChar = (unsigned char*)malloc(width * height * sizeof(unsigned char));

	//test_kernel << <1, 1 >> > ();
	hipEvent_t startMemoryEvent, StopMemoryEvent, startKernelEvent, StopKernelEvent, startMemoryBackEvent, StopMemoryBackEvent;

	//Allocating and copia memoria a la gpu
	hipEventCreate(&startMemoryEvent);
	hipEventCreate(&StopMemoryEvent);
	hipEventRecord(startMemoryEvent);
	float* grayscaledImageDevice; 
	float* FinalImageDevice; 
	hipMalloc((float**)&grayscaledImageDevice, width*height*sizeof(float));
	hipMalloc((float**)&FinalImageDevice, width * height * sizeof(float));
	hipMemcpy(grayscaledImageDevice, grayscaledImage, width * height * sizeof(float), hipMemcpyHostToDevice);
	hipEventRecord(StopMemoryEvent);

	
	//Execution of the kernel
	hipEventCreate(&startKernelEvent);
	hipEventCreate(&StopKernelEvent);
	hipEventRecord(startKernelEvent);
	int tamanyMEITATFinestra = tamanyFinestra / 2; 
	NickKernelMethod1 << <dimGrid, dimBlock >> > (grayscaledImageDevice, FinalImageDevice, k, width, height, tamanyMEITATFinestra);
	//hipDeviceSynchronize();  // SA DE TREURE
	hipEventRecord(StopKernelEvent);


	//Memory Back
	hipEventCreate(&startMemoryBackEvent);
	hipEventCreate(&StopMemoryBackEvent);
	hipEventRecord(startMemoryBackEvent);
	hipMemcpy(FinalImageHost, FinalImageDevice, width * height * sizeof(float), hipMemcpyDeviceToHost);
	hipEventRecord(StopMemoryBackEvent);


	//Free memory, events
	hipFree(grayscaledImageDevice);
	hipFree(FinalImageDevice); 

	hipEventSynchronize(StopMemoryEvent);
	hipEventElapsedTime(&millisecondsMemoryEvent, startMemoryEvent, StopMemoryEvent);

	hipEventSynchronize(StopKernelEvent);
	hipEventElapsedTime(&millisecondsKernelEvent, startKernelEvent, StopKernelEvent);

	hipEventSynchronize(StopMemoryBackEvent);
	hipEventElapsedTime(&millisecondsMemoryBackEvent, startMemoryBackEvent, StopMemoryBackEvent);

	hipEventDestroy(startMemoryEvent);
	hipEventDestroy(StopMemoryEvent);
	hipEventDestroy(startKernelEvent);
	hipEventDestroy(StopKernelEvent);
	hipEventDestroy(startMemoryBackEvent);
	hipEventDestroy(StopMemoryBackEvent);



	//Writing results, and writing file
	hipDeviceSynchronize(); 
	outputDisplay->append("GPU computation done: ");
	outputDisplay->append(QString::fromStdString(string("TIME SPENT ALLOCATING AND COPYING INTO GPU = " + to_string((millisecondsMemoryEvent) / 1000.0f) + " [seconds]" )));
	outputDisplay->append(QString::fromStdString(string("TIME SPENT IN THE GPU KERNEL = " + to_string((millisecondsKernelEvent) / 1000.0) + " [seconds]")));
	outputDisplay->append(QString::fromStdString(string("TIME SPENT COPYING DATA FROM GPU TO CPU = " + to_string((millisecondsMemoryBackEvent) / 1000.0f) + " [seconds]")));

	chrono::steady_clock::time_point begin;
	chrono::steady_clock::time_point end;

	
	outputDisplay->append("CONVERTING IMAGE FLOAT POINTER TO CHAR POINTER TO WRITE THE IMAGE (CPU)...");
	begin = chrono::steady_clock::now();
	
	for (int i = 0; i < width * height; i++) {
		if (FinalImageHost[i] == 0)
		{
			FinalImageHostChar[i] = 0;
		}
		else
		{
			FinalImageHostChar[i] = 255;
		}
		//cout << FinalImageHost[i] << endl; 
	}
	
	end = chrono::steady_clock::now();
	outputDisplay->append(QString::fromStdString(string("CONVERTED FLOAT POINTER TO CHAR POINTER IN(CPU) = " + to_string((chrono::duration_cast<chrono::microseconds>(end - begin).count()) / 1000000.0f) + " [seconds]")));
	
	/*
	for (int i = 0; i < width * height; i++) {
		cout << FinalImageHost[i] << endl;
	}
	*/
	
	outputDisplay->append("WRITING IMAGE...");
	int pixelWidthOUT = 1;
	begin = chrono::steady_clock::now();
	//ESCRITURA DE LA IMAGEN EN SECUENCIAL
	stbi_write_png(fileOUTGPUMETHOD1NICK.c_str(), width, height, pixelWidthOUT, FinalImageHostChar, 0);
	end = chrono::steady_clock::now();
	outputDisplay->append(QString::fromStdString(string("IMAGE WRITTEN IN = " + to_string((chrono::duration_cast<chrono::microseconds>(end - begin).count()) / 1000000.0f) + " [seconds]")));
	outputDisplay->append(QString::fromStdString(string("Nick gpu method1 image saved in: " + fileOUTGPUMETHOD1NICK)));


	//free mmemory, host
	free(FinalImageHostChar);
	free(FinalImageHost);



	return fileOUTGPUMETHOD1NICK;
}





