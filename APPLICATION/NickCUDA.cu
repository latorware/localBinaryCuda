#include "hip/hip_runtime.h"
#ifndef __HIPCC__  
#define __HIPCC__
#endif
#include <iostream>
#include <QTextBrowser>
#include <hip/driver_types.h>
#include <hip/hip_runtime.h>
using namespace std;


#include "./Lib/stb_image_write.h"

#include <malloc.h>
#include <>


#define BLOCKSIZE 32 //32x32 = 1024 (max threads per block)

//max image size: 10.000 x 10.000 pixels
//__constant__ float GrayscaledImageConstant[10000 * 10000];


/*
// gpu kernel function
__global__ void test_kernel(void) {
}
*/

/*
extern "C"
void wrapper(QTextBrowser * outputDisplay)
{
	test_kernel << <1, 1 >> > ();
}
*/



__global__ void NickKernelMethod1(float* grayscaledImageDevice, float* FinalImageDevice, float k, int width, int height, int tamanyMEITATFinestra)
{
	//identificadors threads, fila i columna
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int row = (blockIdx.y * BLOCKSIZE) + ty;
	int col = (blockIdx.x * BLOCKSIZE) + tx;

	//bordes de la finestra lliscant
	int beginrow = max(0, row-tamanyMEITATFinestra);
	int begincolumn = max(0, col - tamanyMEITATFinestra);
	int endrow = min(height-1, row + tamanyMEITATFinestra); 
	int endcolumn = min(width-1, col + tamanyMEITATFinestra); 


	//calcular el pixel actual
	int numeropixelsfinestra = (endrow - beginrow + 1) * (endcolumn - begincolumn + 1);

	if (row < height && col < width)
	{
		float temp;
		float Total_sum = 0;
		float Total_sum_pow2 = 0;
		for (int i = beginrow; i <= endrow; i = i + 1)
			for (int j = begincolumn; j <= endcolumn; j = j + 1)
			{
				temp = grayscaledImageDevice[i * width + j]; 
				//printf("%f \n", temp);
				Total_sum = Total_sum + temp;
				Total_sum_pow2 = Total_sum_pow2 + (temp * temp);
			}

		//printf("%f \n", Total_sum);
		float mean = Total_sum / numeropixelsfinestra;
		float Threshold = mean + k * sqrtf((Total_sum_pow2 - mean * mean) / numeropixelsfinestra);
		//printf("%f      %f \n", Threshold, grayscaledImageDevice[row * width + col]);

		if (Threshold < grayscaledImageDevice[row * width + col])
		{
			FinalImageDevice[row * width + col] = 1; 
			//printf("Yes \n"); 
		}
		else
		{
			FinalImageDevice[row * width + col] = 0; 
			//printf("No \n"); 
		}
		//printf("%f \n", FinalImageDevice[row * width + col]);
	}


}


__global__ void NickKernelMethod2(float* grayscaledImageDevice, float* FinalImageDevice, float k, int width, int height, int tamanyMEITATFinestra)
{
	//Shared   memory
	__shared__ float SharedMemory[BLOCKSIZE][BLOCKSIZE];


	//identificadors threads, fila i columna
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int row = (blockIdx.y * BLOCKSIZE) + ty;
	int col = (blockIdx.x * BLOCKSIZE) + tx;

	//Bordes shared memory
	int beginrowshared = blockIdx.y * BLOCKSIZE;
	int endrowshared = (blockIdx.y + 1) * BLOCKSIZE - 1;
	int begincolshared = blockIdx.x * BLOCKSIZE;
	int endcolshared = (blockIdx.x + 1) * BLOCKSIZE - 1;

	//Carreguem a shared. Cada thread carrega un pixel
	if ((row < height) && (col < width))
		SharedMemory[ty][tx] = grayscaledImageDevice[row * width + col];
	//else
		//SharedMemory[ty][tx] = 0.0f; 
	

	//Ens assegurem que a partir d'aqu� tinguem shared memory carregada
	__syncthreads();


	//bordes de la finestra lliscant
	int beginrow = max(0, row - tamanyMEITATFinestra);
	int begincolumn = max(0, col - tamanyMEITATFinestra);
	int endrow = min(height - 1, row + tamanyMEITATFinestra);
	int endcolumn = min(width - 1, col + tamanyMEITATFinestra);


	//calcular el pixel actual
	int numeropixelsfinestra = (endrow - beginrow + 1) * (endcolumn - begincolumn + 1);

	if (row < height && col < width)
	{
		float temp;
		float Total_sum = 0;
		float Total_sum_pow2 = 0;
		for (int i = beginrow; i <= endrow; i = i + 1)
			for (int j = begincolumn; j <= endcolumn; j = j + 1)
			{
				if ((beginrowshared < i) && (begincolshared < j) && (i < endrowshared) && (j < endcolshared))
					temp = SharedMemory[i % BLOCKSIZE][j % BLOCKSIZE];
				else
					temp = grayscaledImageDevice[i * width + j];
				//printf("%f \n", temp);
				Total_sum = Total_sum + temp;
				Total_sum_pow2 = Total_sum_pow2 + (temp * temp);
			}

		//printf("%f \n", Total_sum);
		float mean = Total_sum / numeropixelsfinestra;
		float Threshold = mean + k * sqrtf((Total_sum_pow2 - mean * mean) / numeropixelsfinestra);
		//printf("%f      %f \n", Threshold, grayscaledImageDevice[row * width + col]);

		if (Threshold < grayscaledImageDevice[row * width + col])
		{
			FinalImageDevice[row * width + col] = 1;
			//printf("Yes \n"); 
		}
		else
		{
			FinalImageDevice[row * width + col] = 0;
			//printf("No \n"); 
		}
		//printf("%f \n", FinalImageDevice[row * width + col]);
	}
}



__global__ void NickKernelMethod3(float* grayscaledImageDevice, float* FinalImageDevice, float k, int width, int height, int tamanyMEITATFinestra)
{
	//Shared   memory
	__shared__ float SharedMemory[BLOCKSIZE*3][BLOCKSIZE*3];


	//identificadors threads, fila i columna
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int row = (blockIdx.y * BLOCKSIZE) + ty;
	int col = (blockIdx.x * BLOCKSIZE) + tx;

	//Bordes shared memory
	int beginrowshared = (blockIdx.y-1) * BLOCKSIZE;
	int endrowshared = (blockIdx.y + 2) * BLOCKSIZE - 1;
	int begincolshared = (blockIdx.x-1) * BLOCKSIZE;
	int endcolshared = (blockIdx.x + 2) * BLOCKSIZE - 1;

	//Carreguem a shared. Cada thread carrega 9 pixels
	if ((row < height) && (col < width))
	{
		//Casos row i col anteriors
		if ((blockIdx.y > 0) && (blockIdx.x > 0))
		{
			//cas -1 -1
			SharedMemory[ty][tx] = grayscaledImageDevice[(beginrowshared + ty) * width + (begincolshared + tx)];
			//cas -1 0
			SharedMemory[ty][tx + BLOCKSIZE] = grayscaledImageDevice[(beginrowshared + ty) * width + col];
			//cas 0 -1
			SharedMemory[ty + BLOCKSIZE][tx] = grayscaledImageDevice[row * width + (begincolshared + tx)];
		}
		else if (blockIdx.y > 0)
		{
			//cas -1 0
			SharedMemory[ty][tx + BLOCKSIZE] = grayscaledImageDevice[(beginrowshared + ty) * width + col];
		}
		else if (blockIdx.x > 0)
		{
			//cas 0 -1
			SharedMemory[ty + BLOCKSIZE][tx] = grayscaledImageDevice[row * width + (begincolshared + tx)];
		}

	
		//La del centre
		SharedMemory[ty + BLOCKSIZE][tx + BLOCKSIZE] = grayscaledImageDevice[row * width + col];



		//Casos row i col seguents
		if ((endrowshared < (height-1)) && (endcolshared < (width-1)))
		{
			//cas 1 1
			SharedMemory[ty + (BLOCKSIZE * 2)][tx + (BLOCKSIZE * 2)] = grayscaledImageDevice[(((blockIdx.y + 1) * BLOCKSIZE) + ty) * width + (((blockIdx.x + 1) * BLOCKSIZE) + tx)];
			//cas 1 0
			SharedMemory[ty + (BLOCKSIZE * 2)][tx + BLOCKSIZE] = grayscaledImageDevice[(((blockIdx.y + 1) * BLOCKSIZE) + ty) * width + col];
			//cas 0 1
			SharedMemory[ty + BLOCKSIZE][tx + (BLOCKSIZE * 2)] = grayscaledImageDevice[row * width + (((blockIdx.x + 1) * BLOCKSIZE) + tx)];
		}
		else if (endrowshared < (height - 1))
		{
			//cas 1 0
			SharedMemory[ty + (BLOCKSIZE * 2)][tx + BLOCKSIZE] = grayscaledImageDevice[(((blockIdx.y + 1) * BLOCKSIZE) + ty) * width + col];
		}
		else if (endcolshared < (width - 1))
		{
			//cas 0 1
			SharedMemory[ty + BLOCKSIZE][tx + (BLOCKSIZE * 2)] = grayscaledImageDevice[row * width + (((blockIdx.x + 1) * BLOCKSIZE) + tx)];
		}

		//Casos 1,-1
		if ((endrowshared < (height - 1)) && (blockIdx.x > 0))
		{
			SharedMemory[ty + (BLOCKSIZE*2)][tx] = grayscaledImageDevice[(((blockIdx.y + 1) * BLOCKSIZE) + ty) * width + (begincolshared + tx)];
		}

		//Casos -1, 1
		if ((blockIdx.y > 0) && endcolshared < (width - 1))
		{
			SharedMemory[ty ][tx + (BLOCKSIZE * 2)] = grayscaledImageDevice[(beginrowshared + ty) * width + (((blockIdx.x + 1) * BLOCKSIZE) + tx)];
		}



		}
	//else
		//SharedMemory[ty][tx] = 0.0f; 


	//Ens assegurem que a partir d'aqu� tinguem shared memory carregada
	__syncthreads();


	//bordes de la finestra lliscant
	int beginrow = max(0, row - tamanyMEITATFinestra);
	int begincolumn = max(0, col - tamanyMEITATFinestra);
	int endrow = min(height - 1, row + tamanyMEITATFinestra);
	int endcolumn = min(width - 1, col + tamanyMEITATFinestra);


	//calcular el pixel actual
	int numeropixelsfinestra = (endrow - beginrow + 1) * (endcolumn - begincolumn + 1);

	if (row < height && col < width)
	{
		float temp;
		float Total_sum = 0;
		float Total_sum_pow2 = 0;
		for (int i = beginrow; i <= endrow; i = i + 1)
			for (int j = begincolumn; j <= endcolumn; j = j + 1)
			{
				if ((beginrowshared < i) && (begincolshared < j) && (i < endrowshared) && (j < endcolshared))
				{
					int indexSharedy = ((((i / BLOCKSIZE) - blockIdx.y) + 1) * BLOCKSIZE) + (i % BLOCKSIZE);
					int indexSharedx = ((((j / BLOCKSIZE) - blockIdx.x) + 1) * BLOCKSIZE) + (j % BLOCKSIZE);
					temp = SharedMemory[indexSharedy][indexSharedx];
				}
				else
					temp = grayscaledImageDevice[i * width + j];
				//printf("%f \n", temp);
				Total_sum = Total_sum + temp;
				Total_sum_pow2 = Total_sum_pow2 + (temp * temp);
			}

		//printf("%f \n", Total_sum);
		float mean = Total_sum / numeropixelsfinestra;
		float Threshold = mean + k * sqrtf((Total_sum_pow2 - mean * mean) / numeropixelsfinestra);
		//printf("%f      %f \n", Threshold, grayscaledImageDevice[row * width + col]);

		if (Threshold < grayscaledImageDevice[row * width + col])
		{
			FinalImageDevice[row * width + col] = 1;
			//printf("Yes \n"); 
		}
		else
		{
			FinalImageDevice[row * width + col] = 0;
			//printf("No \n"); 
		}
		//printf("%f \n", FinalImageDevice[row * width + col]);
	}
}







extern "C"
//Nomes funciona amb memoria global (transicions CPU-GPU amb pinned)
string NICKGPUMethod1(const float* grayscaledImage, int tamanyFinestra, float k, int width, int height, QTextBrowser * outputDisplay, string fileOUTGPUMETHOD1NICK)
{
	float millisecondsMemoryEvent = 0;
	float millisecondsKernelEvent = 0;
	float millisecondsMemoryBackEvent = 0;
	dim3 dimGrid, dimBlock; 

	dimBlock.x = BLOCKSIZE;
	dimBlock.y = BLOCKSIZE;
	dimBlock.z = 1;
	// + BLOCKSIZE necessari pels pixels que queden
	dimGrid.x = (width + BLOCKSIZE - 1) / BLOCKSIZE;
	dimGrid.y = (height + BLOCKSIZE - 1) / BLOCKSIZE;
	dimGrid.z = 1;

	float* FinalImageHost; 
	float* grayscaledImagePinned; 
	unsigned char* FinalImageHostChar = (unsigned char*)malloc(width * height * sizeof(unsigned char));

	hipHostMalloc((float**)&FinalImageHost, width * height * sizeof(float));
	hipHostMalloc((float**)&grayscaledImagePinned, width * height * sizeof(float));
	memcpy(grayscaledImagePinned, grayscaledImage, width * height * sizeof(float)); 

	//test_kernel << <1, 1 >> > ();
	hipEvent_t startMemoryEvent, StopMemoryEvent, startKernelEvent, StopKernelEvent, startMemoryBackEvent, StopMemoryBackEvent;

	//Allocating and copia memoria a la gpu
	hipEventCreate(&startMemoryEvent);
	hipEventCreate(&StopMemoryEvent);
	hipEventRecord(startMemoryEvent);
	float* grayscaledImageDevice; 
	float* FinalImageDevice; 
	hipMalloc((float**)&grayscaledImageDevice, width*height*sizeof(float));
	hipMalloc((float**)&FinalImageDevice, width * height * sizeof(float));
	hipMemcpy(grayscaledImageDevice, grayscaledImagePinned, width * height * sizeof(float), hipMemcpyHostToDevice);
	hipEventRecord(StopMemoryEvent);

	
	//Execution of the kernel
	hipEventCreate(&startKernelEvent);
	hipEventCreate(&StopKernelEvent);
	hipEventRecord(startKernelEvent);
	int tamanyMEITATFinestra = tamanyFinestra / 2; 
	NickKernelMethod1 << <dimGrid, dimBlock >> > (grayscaledImageDevice, FinalImageDevice, k, width, height, tamanyMEITATFinestra);
	//hipDeviceSynchronize();  // SA DE TREURE
	hipEventRecord(StopKernelEvent);


	//Memory Back
	hipEventCreate(&startMemoryBackEvent);
	hipEventCreate(&StopMemoryBackEvent);
	hipEventRecord(startMemoryBackEvent);
	hipMemcpy(FinalImageHost, FinalImageDevice, width * height * sizeof(float), hipMemcpyDeviceToHost);
	hipEventRecord(StopMemoryBackEvent);


	//Free memory, events
	hipFree(grayscaledImageDevice);
	hipFree(FinalImageDevice); 

	hipEventSynchronize(StopMemoryEvent);
	hipEventElapsedTime(&millisecondsMemoryEvent, startMemoryEvent, StopMemoryEvent);

	hipEventSynchronize(StopKernelEvent);
	hipEventElapsedTime(&millisecondsKernelEvent, startKernelEvent, StopKernelEvent);

	hipEventSynchronize(StopMemoryBackEvent);
	hipEventElapsedTime(&millisecondsMemoryBackEvent, startMemoryBackEvent, StopMemoryBackEvent);

	hipEventDestroy(startMemoryEvent);
	hipEventDestroy(StopMemoryEvent);
	hipEventDestroy(startKernelEvent);
	hipEventDestroy(StopKernelEvent);
	hipEventDestroy(startMemoryBackEvent);
	hipEventDestroy(StopMemoryBackEvent);



	//Writing results, and writing file
	hipDeviceSynchronize(); 
	outputDisplay->append("GPU computation done: ");
	outputDisplay->append(QString::fromStdString(string("TIME SPENT ALLOCATING AND COPYING INTO GPU = " + to_string((millisecondsMemoryEvent) / 1000.0f) + " [seconds]" )));
	outputDisplay->append(QString::fromStdString(string("TIME SPENT IN THE GPU KERNEL = " + to_string((millisecondsKernelEvent) / 1000.0) + " [seconds]")));
	outputDisplay->append(QString::fromStdString(string("TIME SPENT COPYING DATA FROM GPU TO CPU = " + to_string((millisecondsMemoryBackEvent) / 1000.0f) + " [seconds]")));

	chrono::steady_clock::time_point begin;
	chrono::steady_clock::time_point end;

	
	outputDisplay->append("CONVERTING IMAGE FLOAT POINTER TO CHAR POINTER TO WRITE THE IMAGE (CPU)...");
	begin = chrono::steady_clock::now();
	
	for (int i = 0; i < width * height; i++) {
		if (FinalImageHost[i] == 0)
		{
			FinalImageHostChar[i] = 0;
		}
		else
		{
			FinalImageHostChar[i] = 255;
		}
		//cout << FinalImageHost[i] << endl; 
	}
	
	end = chrono::steady_clock::now();
	outputDisplay->append(QString::fromStdString(string("CONVERTED FLOAT POINTER TO CHAR POINTER IN(CPU) = " + to_string((chrono::duration_cast<chrono::microseconds>(end - begin).count()) / 1000000.0f) + " [seconds]")));
	
	/*
	for (int i = 0; i < width * height; i++) {
		cout << FinalImageHost[i] << endl;
	}
	*/
	
	outputDisplay->append("WRITING IMAGE...");
	int pixelWidthOUT = 1;
	begin = chrono::steady_clock::now();
	//ESCRITURA DE LA IMAGEN EN SECUENCIAL
	stbi_write_png(fileOUTGPUMETHOD1NICK.c_str(), width, height, pixelWidthOUT, FinalImageHostChar, 0);
	end = chrono::steady_clock::now();
	outputDisplay->append(QString::fromStdString(string("IMAGE WRITTEN IN = " + to_string((chrono::duration_cast<chrono::microseconds>(end - begin).count()) / 1000000.0f) + " [seconds]")));
	outputDisplay->append(QString::fromStdString(string("Nick gpu method1 image saved in: " + fileOUTGPUMETHOD1NICK)));


	//free mmemory, host
	
	free(FinalImageHostChar);
	hipHostFree(FinalImageHost);
	hipHostFree(grayscaledImagePinned); 



	return fileOUTGPUMETHOD1NICK;
}

//Funciona amb shared i global (transicions CPU-GPU amb pinned)
extern "C"
string NICKGPUMethod2(const float* grayscaledImage, int tamanyFinestra, float k, int width, int height, QTextBrowser* outputDisplay, string fileOUTGPUMETHOD2NICK)
{
	float millisecondsMemoryEvent = 0;
	float millisecondsKernelEvent = 0;
	float millisecondsMemoryBackEvent = 0;
	dim3 dimGrid, dimBlock;

	dimBlock.x = BLOCKSIZE;
	dimBlock.y = BLOCKSIZE;
	dimBlock.z = 1;
	// + BLOCKSIZE necessari pels pixels que queden
	dimGrid.x = (width + BLOCKSIZE - 1) / BLOCKSIZE;
	dimGrid.y = (height + BLOCKSIZE - 1) / BLOCKSIZE;
	dimGrid.z = 1;

	float* FinalImageHost;
	float* grayscaledImagePinned;
	unsigned char* FinalImageHostChar = (unsigned char*)malloc(width * height * sizeof(unsigned char));

	hipHostMalloc((float**)&FinalImageHost, width * height * sizeof(float));
	hipHostMalloc((float**)&grayscaledImagePinned, width * height * sizeof(float));
	memcpy(grayscaledImagePinned, grayscaledImage, width * height * sizeof(float));

	//test_kernel << <1, 1 >> > ();
	hipEvent_t startMemoryEvent, StopMemoryEvent, startKernelEvent, StopKernelEvent, startMemoryBackEvent, StopMemoryBackEvent;

	//Allocating and copia memoria a la gpu
	hipEventCreate(&startMemoryEvent);
	hipEventCreate(&StopMemoryEvent);
	hipEventRecord(startMemoryEvent);
	float* grayscaledImageDevice;
	float* FinalImageDevice;
	hipMalloc((float**)&grayscaledImageDevice, width * height * sizeof(float));
	hipMalloc((float**)&FinalImageDevice, width * height * sizeof(float));
	hipMemcpy(grayscaledImageDevice, grayscaledImagePinned, width * height * sizeof(float), hipMemcpyHostToDevice);
	hipEventRecord(StopMemoryEvent);


	//Execution of the kernel
	hipEventCreate(&startKernelEvent);
	hipEventCreate(&StopKernelEvent);
	hipEventRecord(startKernelEvent);
	int tamanyMEITATFinestra = tamanyFinestra / 2;
	NickKernelMethod2 << <dimGrid, dimBlock >> > (grayscaledImageDevice, FinalImageDevice, k, width, height, tamanyMEITATFinestra);
	//hipDeviceSynchronize();  // SA DE TREURE
	hipEventRecord(StopKernelEvent);


	//Memory Back
	hipEventCreate(&startMemoryBackEvent);
	hipEventCreate(&StopMemoryBackEvent);
	hipEventRecord(startMemoryBackEvent);
	hipMemcpy(FinalImageHost, FinalImageDevice, width * height * sizeof(float), hipMemcpyDeviceToHost);
	hipEventRecord(StopMemoryBackEvent);


	//Free memory, events
	hipFree(grayscaledImageDevice);
	hipFree(FinalImageDevice);

	hipEventSynchronize(StopMemoryEvent);
	hipEventElapsedTime(&millisecondsMemoryEvent, startMemoryEvent, StopMemoryEvent);

	hipEventSynchronize(StopKernelEvent);
	hipEventElapsedTime(&millisecondsKernelEvent, startKernelEvent, StopKernelEvent);

	hipEventSynchronize(StopMemoryBackEvent);
	hipEventElapsedTime(&millisecondsMemoryBackEvent, startMemoryBackEvent, StopMemoryBackEvent);

	hipEventDestroy(startMemoryEvent);
	hipEventDestroy(StopMemoryEvent);
	hipEventDestroy(startKernelEvent);
	hipEventDestroy(StopKernelEvent);
	hipEventDestroy(startMemoryBackEvent);
	hipEventDestroy(StopMemoryBackEvent);



	//Writing results, and writing file
	hipDeviceSynchronize();
	outputDisplay->append("GPU computation done: ");
	outputDisplay->append(QString::fromStdString(string("TIME SPENT ALLOCATING AND COPYING INTO GPU = " + to_string((millisecondsMemoryEvent) / 1000.0f) + " [seconds]")));
	outputDisplay->append(QString::fromStdString(string("TIME SPENT IN THE GPU KERNEL = " + to_string((millisecondsKernelEvent) / 1000.0) + " [seconds]")));
	outputDisplay->append(QString::fromStdString(string("TIME SPENT COPYING DATA FROM GPU TO CPU = " + to_string((millisecondsMemoryBackEvent) / 1000.0f) + " [seconds]")));

	chrono::steady_clock::time_point begin;
	chrono::steady_clock::time_point end;


	outputDisplay->append("CONVERTING IMAGE FLOAT POINTER TO CHAR POINTER TO WRITE THE IMAGE (CPU)...");
	begin = chrono::steady_clock::now();

	for (int i = 0; i < width * height; i++) {
		if (FinalImageHost[i] == 0)
		{
			FinalImageHostChar[i] = 0;
		}
		else
		{
			FinalImageHostChar[i] = 255;
		}
		//cout << FinalImageHost[i] << endl; 
	}

	end = chrono::steady_clock::now();
	outputDisplay->append(QString::fromStdString(string("CONVERTED FLOAT POINTER TO CHAR POINTER IN(CPU) = " + to_string((chrono::duration_cast<chrono::microseconds>(end - begin).count()) / 1000000.0f) + " [seconds]")));

	/*
	for (int i = 0; i < width * height; i++) {
		cout << FinalImageHost[i] << endl;
	}
	*/

	outputDisplay->append("WRITING IMAGE...");
	int pixelWidthOUT = 1;
	begin = chrono::steady_clock::now();
	//ESCRITURA DE LA IMAGEN EN SECUENCIAL
	stbi_write_png(fileOUTGPUMETHOD2NICK.c_str(), width, height, pixelWidthOUT, FinalImageHostChar, 0);
	end = chrono::steady_clock::now();
	outputDisplay->append(QString::fromStdString(string("IMAGE WRITTEN IN = " + to_string((chrono::duration_cast<chrono::microseconds>(end - begin).count()) / 1000000.0f) + " [seconds]")));
	outputDisplay->append(QString::fromStdString(string("Nick gpu method2 image saved in: " + fileOUTGPUMETHOD2NICK)));


	//free mmemory, host

	free(FinalImageHostChar);
	hipHostFree(FinalImageHost);
	hipHostFree(grayscaledImagePinned);



	return fileOUTGPUMETHOD2NICK;
}




//Funciona amb shared i global (transicions CPU-GPU amb pinned)
extern "C"
string NICKGPUMethod3(const float* grayscaledImage, int tamanyFinestra, float k, int width, int height, QTextBrowser * outputDisplay, string fileOUTGPUMETHOD3NICK)
{
	float millisecondsMemoryEvent = 0;
	float millisecondsKernelEvent = 0;
	float millisecondsMemoryBackEvent = 0;
	dim3 dimGrid, dimBlock;

	dimBlock.x = BLOCKSIZE;
	dimBlock.y = BLOCKSIZE;
	dimBlock.z = 1;
	// + BLOCKSIZE necessari pels pixels que queden
	dimGrid.x = (width + BLOCKSIZE - 1) / BLOCKSIZE;
	dimGrid.y = (height + BLOCKSIZE - 1) / BLOCKSIZE;
	dimGrid.z = 1;

	float* FinalImageHost;
	float* grayscaledImagePinned;
	unsigned char* FinalImageHostChar = (unsigned char*)malloc(width * height * sizeof(unsigned char));

	hipHostMalloc((float**)&FinalImageHost, width * height * sizeof(float));
	hipHostMalloc((float**)&grayscaledImagePinned, width * height * sizeof(float));
	memcpy(grayscaledImagePinned, grayscaledImage, width * height * sizeof(float));

	//test_kernel << <1, 1 >> > ();
	hipEvent_t startMemoryEvent, StopMemoryEvent, startKernelEvent, StopKernelEvent, startMemoryBackEvent, StopMemoryBackEvent;

	//Allocating and copia memoria a la gpu
	hipEventCreate(&startMemoryEvent);
	hipEventCreate(&StopMemoryEvent);
	hipEventRecord(startMemoryEvent);
	float* grayscaledImageDevice;
	float* FinalImageDevice;
	hipMalloc((float**)&grayscaledImageDevice, width * height * sizeof(float));
	hipMalloc((float**)&FinalImageDevice, width * height * sizeof(float));
	hipMemcpy(grayscaledImageDevice, grayscaledImagePinned, width * height * sizeof(float), hipMemcpyHostToDevice);
	hipEventRecord(StopMemoryEvent);


	//Execution of the kernel
	hipEventCreate(&startKernelEvent);
	hipEventCreate(&StopKernelEvent);
	hipEventRecord(startKernelEvent);
	int tamanyMEITATFinestra = tamanyFinestra / 2;
	NickKernelMethod3 << <dimGrid, dimBlock >> > (grayscaledImageDevice, FinalImageDevice, k, width, height, tamanyMEITATFinestra);
	//hipDeviceSynchronize();  // SA DE TREURE
	hipEventRecord(StopKernelEvent);


	//Memory Back
	hipEventCreate(&startMemoryBackEvent);
	hipEventCreate(&StopMemoryBackEvent);
	hipEventRecord(startMemoryBackEvent);
	hipMemcpy(FinalImageHost, FinalImageDevice, width * height * sizeof(float), hipMemcpyDeviceToHost);
	hipEventRecord(StopMemoryBackEvent);


	//Free memory, events
	hipFree(grayscaledImageDevice);
	hipFree(FinalImageDevice);

	hipEventSynchronize(StopMemoryEvent);
	hipEventElapsedTime(&millisecondsMemoryEvent, startMemoryEvent, StopMemoryEvent);

	hipEventSynchronize(StopKernelEvent);
	hipEventElapsedTime(&millisecondsKernelEvent, startKernelEvent, StopKernelEvent);

	hipEventSynchronize(StopMemoryBackEvent);
	hipEventElapsedTime(&millisecondsMemoryBackEvent, startMemoryBackEvent, StopMemoryBackEvent);

	hipEventDestroy(startMemoryEvent);
	hipEventDestroy(StopMemoryEvent);
	hipEventDestroy(startKernelEvent);
	hipEventDestroy(StopKernelEvent);
	hipEventDestroy(startMemoryBackEvent);
	hipEventDestroy(StopMemoryBackEvent);



	//Writing results, and writing file
	hipDeviceSynchronize();
	outputDisplay->append("GPU computation done: ");
	outputDisplay->append(QString::fromStdString(string("TIME SPENT ALLOCATING AND COPYING INTO GPU = " + to_string((millisecondsMemoryEvent) / 1000.0f) + " [seconds]")));
	outputDisplay->append(QString::fromStdString(string("TIME SPENT IN THE GPU KERNEL = " + to_string((millisecondsKernelEvent) / 1000.0) + " [seconds]")));
	outputDisplay->append(QString::fromStdString(string("TIME SPENT COPYING DATA FROM GPU TO CPU = " + to_string((millisecondsMemoryBackEvent) / 1000.0f) + " [seconds]")));

	chrono::steady_clock::time_point begin;
	chrono::steady_clock::time_point end;


	outputDisplay->append("CONVERTING IMAGE FLOAT POINTER TO CHAR POINTER TO WRITE THE IMAGE (CPU)...");
	begin = chrono::steady_clock::now();

	for (int i = 0; i < width * height; i++) {
		if (FinalImageHost[i] == 0)
		{
			FinalImageHostChar[i] = 0;
		}
		else
		{
			FinalImageHostChar[i] = 255;
		}
		//cout << FinalImageHost[i] << endl; 
	}

	end = chrono::steady_clock::now();
	outputDisplay->append(QString::fromStdString(string("CONVERTED FLOAT POINTER TO CHAR POINTER IN(CPU) = " + to_string((chrono::duration_cast<chrono::microseconds>(end - begin).count()) / 1000000.0f) + " [seconds]")));

	/*
	for (int i = 0; i < width * height; i++) {
		cout << FinalImageHost[i] << endl;
	}
	*/

	outputDisplay->append("WRITING IMAGE...");
	int pixelWidthOUT = 1;
	begin = chrono::steady_clock::now();
	//ESCRITURA DE LA IMAGEN EN SECUENCIAL
	stbi_write_png(fileOUTGPUMETHOD3NICK.c_str(), width, height, pixelWidthOUT, FinalImageHostChar, 0);
	end = chrono::steady_clock::now();
	outputDisplay->append(QString::fromStdString(string("IMAGE WRITTEN IN = " + to_string((chrono::duration_cast<chrono::microseconds>(end - begin).count()) / 1000000.0f) + " [seconds]")));
	outputDisplay->append(QString::fromStdString(string("Nick gpu method3 image saved in: " + fileOUTGPUMETHOD3NICK)));


	//free mmemory, host

	free(FinalImageHostChar);
	hipHostFree(FinalImageHost);
	hipHostFree(grayscaledImagePinned);



	return fileOUTGPUMETHOD3NICK;
}