#include "hip/hip_runtime.h"
#include <iostream>
#include <QTextBrowser>
#include <hip/driver_types.h>
#include <hip/hip_runtime.h>
using namespace std;


#include "./Lib/stb_image_write.h"

#include <malloc.h>


#define BLOCKSIZE 32

//max image size: 10.000 x 10.000 pixels
//__constant__ float GrayscaledImageConstant[10000 * 10000];


/*
// gpu kernel function
__global__ void test_kernel(void) {
}
*/





/*
extern "C"
void wrapper(QTextBrowser * outputDisplay)
{
	test_kernel << <1, 1 >> > ();
}
*/

extern "C"
string NICKGPUMethod1(const float* grayscaledImage, int tamanyFinestra, float k, int width, int height, QTextBrowser * outputDisplay, string fileOUTGPUMETHOD1NICK)
{

	dim3 dimGrid, dimBlock; 

	dimBlock.x = BLOCKSIZE;
	dimBlock.y = BLOCKSIZE;
	dimBlock.z = 1;

	dimGrid.x = (width + BLOCKSIZE - 1) / BLOCKSIZE;
	dimGrid.y = (height + BLOCKSIZE - 1) / BLOCKSIZE;
	dimGrid.z = 1;

	float* FinalImageHost = (float*)malloc(width * height * sizeof(float));

	//test_kernel << <1, 1 >> > ();
	hipEvent_t startMemoryEvent, StopMemoryEvent, startKernelEvent, StopKernelEvent, startMemoryBackEvent, StopMemoryBackEvent;

	//Allocating and copia memoria a la gpu
	hipEventCreate(&startMemoryEvent);
	hipEventCreate(&StopMemoryEvent);
	hipEventRecord(startMemoryEvent);
	float* grayscaledImageDevice; 
	float* FinalImageDevice; 
	hipMalloc((float**)&grayscaledImageDevice, width*height*sizeof(float));
	hipMalloc((float**)&FinalImageDevice, width * height * sizeof(float));
	hipMemcpy(grayscaledImageDevice, grayscaledImage, width * height * sizeof(float), hipMemcpyHostToDevice);
	hipEventRecord(StopMemoryEvent);

	
	//Execution of the kernel
	hipEventCreate(&startKernelEvent);
	hipEventCreate(&StopKernelEvent);
	hipEventRecord(startKernelEvent);
	//NickKernel << <dimGrid, dimBlock >> > (Nd, Pd_global, Nd.width, Nd.height);
	hipEventRecord(StopKernelEvent);


	//Memory Back
	hipEventCreate(&startMemoryBackEvent);
	hipEventCreate(&StopMemoryBackEvent);
	hipEventRecord(startMemoryBackEvent);
	hipMemcpy(FinalImageHost, FinalImageDevice, width * height * sizeof(float), hipMemcpyDeviceToHost);
	hipEventRecord(StopMemoryBackEvent);


	//Free memory, events
	hipFree(grayscaledImageDevice);
	hipFree(FinalImageDevice); 

	hipEventSynchronize(StopMemoryEvent);
	float millisecondsMemoryEvent = 0;
	hipEventElapsedTime(&millisecondsMemoryEvent, startMemoryEvent, StopMemoryEvent);

	hipEventSynchronize(StopKernelEvent);
	float millisecondsKernelEvent = 0;
	hipEventElapsedTime(&millisecondsKernelEvent, startKernelEvent, StopKernelEvent);

	hipEventSynchronize(StopMemoryBackEvent);
	float millisecondsMemoryBackEvent = 0;
	hipEventElapsedTime(&millisecondsMemoryBackEvent, startMemoryBackEvent, StopMemoryBackEvent);

	hipEventDestroy(startMemoryEvent);
	hipEventDestroy(StopMemoryEvent);
	hipEventDestroy(startKernelEvent);
	hipEventDestroy(StopKernelEvent);
	hipEventDestroy(startMemoryBackEvent);
	hipEventDestroy(StopMemoryBackEvent);



	//Writing results, and writing file
	hipDeviceSynchronize(); 
	outputDisplay->append("GPU computation done: ");
	outputDisplay->append(QString::fromStdString(string("TIME SPENT ALLOCATING AND COPYING INTO GPU = " + to_string((millisecondsMemoryEvent) / 1000.0) + " [seconds]" )));
	outputDisplay->append(QString::fromStdString(string("TIME SPENT IN THE GPU KERNEL = " + to_string((millisecondsKernelEvent) / 1000.0) + " [seconds]")));
	outputDisplay->append(QString::fromStdString(string("TIME SPENT COPYING DATA FROM GPU TO CPU = " + to_string((millisecondsMemoryBackEvent) / 1000.0) + " [seconds]")));

	outputDisplay->append("WRITING IMAGE...");
	int pixelWidthOUT = 1;
	chrono::steady_clock::time_point begin;
	chrono::steady_clock::time_point end;
	begin = chrono::steady_clock::now();
	//ESCRITURA DE LA IMAGEN EN SECUENCIAL
	stbi_write_png(fileOUTGPUMETHOD1NICK.c_str(), width, height, pixelWidthOUT, FinalImageDevice, 0);
	end = chrono::steady_clock::now();
	outputDisplay->append(QString::fromStdString(string("IMAGE WRITTEN IN = " + to_string((chrono::duration_cast<chrono::microseconds>(end - begin).count()) / 1000000.0) + " [seconds]")));
	outputDisplay->append(QString::fromStdString(string("Nick gpu method1 image saved in: " + fileOUTGPUMETHOD1NICK)));


	//free mmemory, host
	free(FinalImageDevice); 



	return fileOUTGPUMETHOD1NICK;
}





